#include "hip/hip_runtime.h"
#include <El-lite.hpp>
#include <El/blas_like/level1.hpp>
#include <El/blas_like/level1/GPU/Axpy.hpp>
#include <El/core/imports/cuda.hpp>

namespace
{

template <typename T>
__global__ void Axpy_kernel( size_t height, size_t width,
                             T alpha,
                             T const* X, size_t colStrideX, size_t rowStrideX,
                             T* Y, size_t colStrideY, size_t rowStrideY )
{
    const size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    const size_t numThreads = blockDim.x * gridDim.x;
    for (size_t pos = tid; pos < height * width; pos += numThreads) {
        const size_t i = pos % height;
        const size_t j = pos / height;
        Y[i*colStrideY+j*rowStrideY] += alpha * X[i*colStrideX+j*rowStrideX];
    }
}

}// namespace <anon>

namespace El
{

template <typename T, typename>
void Axpy_GPU_impl(
    size_t height, size_t width,
    T const& alpha,
    T const* X, size_t colStrideX, size_t rowStrideX,
    T* Y, size_t colStrideY, size_t rowStrideY )
{
    if( height <= 0 || width <= 0 ) { return; }
    const size_t size = height * width;
    const size_t blockDim = 256;
    const size_t gridDim = (size + blockDim - 1) / blockDim;
    hipStream_t stream = GPUManager::Stream();
    EL_CHECK_CUDA_KERNEL( Axpy_kernel<T>,
                          gridDim, blockDim, 0, stream,
                          ( height, width, alpha,
                            X, colStrideX, rowStrideX,
                            Y, colStrideY, rowStrideY ) );
}

template void Axpy_GPU_impl(
    size_t, size_t, float const&,
    float const*, size_t, size_t, float*, size_t, size_t);
template void Axpy_GPU_impl(
    size_t, size_t, double const&,
    double const*, size_t, size_t, double*, size_t, size_t);

}// namespace El
